#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

const int nstates = %(NGENERATORS);
__device__ hiprandState_t* states[nstates];

__global__ void initkernel(int seed)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tidx < nstates) {
        hiprandState_t* s = new hiprandState_t;
        if (s != 0) {
            hiprand_init(seed, tidx, 0, s);
        }

        states[tidx] = s;
    }
}

__global__ void randfillkernel(float *values, int N)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tidx < nstates) {
        hiprandState_t s = *states[tidx];
        for(int i=tidx; i < N; i += blockDim.x * gridDim.x) {
            values[i] = hiprand_uniform(&s);
        }
        *states[tidx] = s;
    }
}